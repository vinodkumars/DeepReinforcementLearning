
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>
#include <float.h>

#define IDX2C(i,j,rows) (((j)*(rows))+(i))

__global__ void matrixPlusVector(float* input, float* bias, float * output, int rows, int columns)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i < rows && j < columns)
	{
		int ij = IDX2C(i, j, rows);
		output[ij] = input[ij] + bias[i];
	}
}

__global__ void matrixTanh(float* input, float* output, int rows, int columns)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i < rows && j < columns)
	{
		int ij = IDX2C(i, j, rows);
		output[ij] = tanh(input[ij]);
	}
}

__global__ void matrixIncorporateTanhDeriv(float* base, float* activation, float* output, int rows, int columns)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i < rows && j < columns)
	{
		int ij = IDX2C(i, j, rows);
		output[ij] = base[ij] * (1 + activation[ij])*(1 - activation[ij]);
	}
}

__global__ void matrixReLu(float* input, float* output, int rows, int columns)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i < rows && j < columns)
	{
		int ij = IDX2C(i, j, rows);
		output[ij] = fmaxf(input[ij], 0);
	}
}

__global__ void matrixIncorporateReLuDeriv(float* base, float* activation, float* output, int rows, int columns)
{
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	int i = blockDim.y * blockIdx.y + threadIdx.y;
	if (i < rows && j < columns)
	{
		int ij = IDX2C(i, j, rows);
		output[ij] = activation[ij] <= 0 ? 0 : base[ij];
	}
}

__global__ void matrixSigmoid(float* input, float* output, int rows, int columns)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i < rows && j < columns)
	{
		int ij = IDX2C(i, j, rows);
		// how to refactor the sigmoid calculation???
		output[ij] = (tanhf((input[ij]) / 2) + 1) / 2.0f;
	}
}

__global__ void matrixIncorporateSigmoidDeriv(float* base, float* activation, float* output, int rows, int columns)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i < rows && j < columns)
	{
		int ij = IDX2C(i, j, rows);
		output[ij] = base[ij] * activation[ij] * (1 - activation[ij]);
	}
}

__global__ void matrixCrossEntropyError(float* sigmoidScores, float* trueLabels, float* output, int rows, int columns)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i < rows && j < columns)
	{
		int ij = IDX2C(i, j, rows);
		output[ij] = trueLabels[ij] > 0 ? logf(sigmoidScores[ij] + FLT_EPSILON) : logf(1 - sigmoidScores[ij] + FLT_EPSILON);
		output[ij] *= -1;
	}
}

__global__ void matrixBellmanErrorAndDeriv(float* predictedQValues, float* maxQHatValues, float* chosenActionIndices, float* currentRewards, float* error, float* errorDerivative,
	float discount, float* isLastEpisode, int rows, int columns)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i < rows && j < columns)
	{
		int ij = IDX2C(i, j, rows);
		float y = isLastEpisode[j] > 0 ? currentRewards[j] : currentRewards[j] + (discount*maxQHatValues[j]);
		errorDerivative[ij] = 0;

		// Calculating error and errorDerivative
		if (i == chosenActionIndices[j])
		{
			float tmp = predictedQValues[i] - y;
			errorDerivative[ij] = tmp;
			error[j] = 0.5*tmp*tmp;
		}
	}
}

__global__ void DqnStanfordEvaluation(float* predictedactionIndices, float* chosenActionIndices, float* currentRewards, float* matchPredictRewards, float* nonMatchPredictRewards, int rows)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < rows)
	{
		if (predictedactionIndices[i] == chosenActionIndices[i])
		{
			matchPredictRewards[i] = currentRewards[i];
		}
		else
		{
			nonMatchPredictRewards[i] = currentRewards[i];
		}
	}
}

__global__ void matrixHadamard(float* input1, float* input2, float alpha, float* output, float beta, int rows, int columns)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i < rows && j < columns)
	{
		int ij = IDX2C(i, j, rows);
		output[ij] = alpha*input1[ij] * input2[ij] + beta*output[ij];
	}
}

__global__ void columnwiseMax(float* input, float* output, int rows, int columns)
{
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	if (j < columns)
	{
		float maxInColumn = input[IDX2C(0, j, rows)];
		for (int i = 0; i < rows; i++)
		{
			int ij = IDX2C(i, j, rows);
			if (input[ij] > maxInColumn)
			{
				maxInColumn = input[ij];
			}
		}
		output[j] = maxInColumn;
	}
}

__global__ void columnwiseMaxIndex(float* input, float* output, int rows, int columns)
{
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	if (j < columns)
	{
		int maxInColumnIndex = 0;
		float maxInColumn = input[IDX2C(maxInColumnIndex, j, rows)];
		for (int i = 0; i < rows; i++)
		{
			int ij = IDX2C(i, j, rows);
			if (input[ij] > maxInColumn)
			{
				maxInColumn = input[ij];
				maxInColumnIndex = i;
			}
		}
		output[j] = (float)maxInColumnIndex;
	}
}

int main()
{
	return 0;
}